#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <vector>
#include <stdlib.h>
#include <stddef.h>
#include "CudaArray.h"
#include "Matrix_Operations.h"

using namespace std;

__global__ void matrixSubtractionKernel(double* A, double* B, double* C, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows && j < cols) {
        C[i * cols + j] = A[i * cols + j] - B[i * cols + j];
    }
}


void internalMatrixSubtract(double* A, double* B, double* C, int rows, int cols) {

    // declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block
    dim3 threadsPerBlock(rows, cols);
    dim3 blocksPerGrid(1, 1);

    if (rows > 32) {
        threadsPerBlock.x = 32;
        blocksPerGrid.x = static_cast<int>(ceil(double(rows) / double(threadsPerBlock.x)));
    }

    if (cols > 32) {
        threadsPerBlock.y = 32;
        blocksPerGrid.y = static_cast<int>(ceil(double(cols) / double(threadsPerBlock.y)));
    }

    matrixSubtractionKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, rows, cols);
    hipDeviceSynchronize();
}

void matrix_subtract(const vector<double>& A, const vector<double>& B, vector<double>& C, int Arows, int Acols) {

    CudaArray<double> d_A(A.size());
    CudaArray<double> d_B(B.size());
    CudaArray<double> d_C(C.size());

    d_A.set(A);
    d_B.set(B);

    internalMatrixSubtract(d_A.getData(), d_B.getData(), d_C.getData(), Arows, Acols);
    d_C.get(C.data(), C.size());
}
