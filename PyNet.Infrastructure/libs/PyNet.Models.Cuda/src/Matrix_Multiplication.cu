#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <cmath>
#include "CudaArray.h"
#include <vector>
#include "Matrix_Operations.h"

using namespace std;

__global__ void matrixMultiplicationKernel(double* A, double* B, double* C, int Arows, int Acols, int Bcols) {
    int ROW = blockIdx.x * blockDim.x + threadIdx.x;
    int COL = blockIdx.y * blockDim.y + threadIdx.y;

    double tmpSum = 0;

    if (ROW < Arows && COL < Bcols) {
        for (auto i = 0; i < Acols; i++) {
            tmpSum += A[ROW * Acols + i] * B[i * Bcols + COL];
        }

        C[ROW * Bcols + COL] = tmpSum;
    }
}

void internalMatrixMultiply(double* A, double* B, double* C, int Arows, int Acols, int Bcols) {

    // declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block
    dim3 threadsPerBlock(Arows, Bcols);
    dim3 blocksPerGrid(1, 1);
    if (Arows > 32) {
        threadsPerBlock.x = 32;
        blocksPerGrid.x = static_cast<int>(ceil(double(Arows) / double(threadsPerBlock.x)));
    }

    if (Bcols > 32) {
        threadsPerBlock.y = 32;
        blocksPerGrid.y = static_cast<int>(ceil(double(Bcols) / double(threadsPerBlock.y)));
    }

    matrixMultiplicationKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, Arows, Acols, Bcols);
    hipDeviceSynchronize();
}

void matrix_multiply(const vector<double>& A, const vector<double>& B, vector<double>& C, int Arows, int Acols, int Bcols) {

    CudaArray<double> d_A(A.size());
    CudaArray<double> d_B(B.size());
    CudaArray<double> d_C(C.size());

    d_A.set(A);
    d_B.set(B);

    internalMatrixMultiply(d_A.getData(), d_B.getData(), d_C.getData(), Arows, Acols, Bcols);
    d_C.get(C.data(), C.size());
}