#include "hip/hip_runtime.h"
﻿#include <math.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "cuda_array.h"
#include <vector>
#include <stdlib.h>
#include <stddef.h>
#include "Matrix_Operations.h"

__global__ void matrixMultiplicationKernel(double* A, double* B, double* C, int Arows, int Bcols) {
    int ROW = blockIdx.x * blockDim.x + threadIdx.x;
    int COL = blockIdx.y * blockDim.y + threadIdx.y;

    double tmpSum = 0;

    if (ROW < Arows && COL < Bcols) {
        // each thread computes one element of the block sub-matrix
        for (auto i = 0; i < Arows; i++) {
            tmpSum += A[ROW * Arows + i] * B[i * Bcols + COL];
        }

        C[ROW * Bcols + COL] = tmpSum;
    }
}

__global__ void matrixDoubleMultiplicationKernel(double* A, double* B, double* C, int Acols, int Arows) {
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;

    if (ROW < Acols && COL < Arows) {
        C[ROW * Acols + COL] = C[ROW * Acols + COL] * *B;
    }
}

void internalMatrixMultiply(double* A, double* B, double* C, int Arows, int Bcols) {

    // declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block
    dim3 threadsPerBlock(Arows, Bcols);
    dim3 blocksPerGrid(1, 1);
    if (Arows * Bcols > 512) {
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
        blocksPerGrid.x = static_cast<int>(ceil(double(Arows) / double(threadsPerBlock.x)));
        blocksPerGrid.y = static_cast<int>(ceil(double(Bcols) / double(threadsPerBlock.y)));
    }

    matrixMultiplicationKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, Arows, Bcols);
    hipDeviceSynchronize();
}

void internalMatrixDoubleMultiply(double* A, double* B, double* C, int Acols, int Arows) {

    // declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block
    dim3 threadsPerBlock(Acols, Arows);
    dim3 blocksPerGrid(1, 1);
    if (Acols * Arows > 512) {
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
        blocksPerGrid.x = static_cast<int>(ceil(double(Acols) / double(threadsPerBlock.x)));
        blocksPerGrid.y = static_cast<int>(ceil(double(Arows) / double(threadsPerBlock.y)));
    }

    matrixMultiplicationKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, Acols, Arows);
    hipDeviceSynchronize();
}

void cuda_matrix_multiply(const Matrix& A, const Matrix& B, Matrix& C) {

    cuda_array<double> d_A(A.GetCValues().size());
    cuda_array<double> d_B(B.GetCValues().size());
    cuda_array<double> d_C(C.GetCValues().size());

    d_A.set(A.GetCValues());
    d_B.set(B.GetCValues());

    internalMatrixMultiply(d_A.getData(), d_B.getData(), d_C.getData(), A.GetRows(), B.GetCols());

    d_C.get(C.GetValues().data(), C.GetSize());
}

void multiply_matrix_and_double(const Matrix& A, const double B, Matrix& C) {

    cuda_array<double> d_A(A.GetCValues().size());
    cuda_array<double> d_B(1);
    cuda_array<double> d_C(C.GetCValues().size());

    d_A.set(A.GetCValues());

    std::vector<double> bVector{ B };
    d_B.set(bVector);

    internalMatrixMultiply(d_A.getData(), d_B.getData(), d_C.getData(), A.GetCols(), A.GetRows());
    d_C.get(C.GetValues().data(), C.GetSize());
}
