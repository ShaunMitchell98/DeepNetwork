#include "hip/hip_runtime.h"
﻿#include <math.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "cuda_array.h"
#include <vector>
#include <stdlib.h>
#include <stddef.h>
#include "Matrix_Operations.h"

__global__ void matrixMultiplicationKernel(double* A, double* B, double* C, int Acols, int Bcols) {
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;

    double tmpSum = 0;

    if (ROW < Acols && COL < Bcols) {
        // each thread computes one element of the block sub-matrix
        for (auto i = 0; i < Acols; i++) {
            tmpSum += A[ROW * Acols + i] * B[i * Bcols + COL];
        }

        C[ROW * Bcols + COL] = tmpSum;
    }
}

__global__ void matrixDoubleMultiplicationKernel(double* A, double* B, double* C, int Acols, int Arows) {
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;

    if (ROW < Acols && COL < Arows) {
        C[ROW * Acols + COL] = C[ROW * Acols + COL] * *B;
    }
}

void internalMatrixMultiply(double* A, double* B, double* C, int Acols, int Bcols) {

    // declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block
    dim3 threadsPerBlock(Acols, Acols);
    dim3 blocksPerGrid(1, 1);
    if (Acols * Acols > 512) {
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
        blocksPerGrid.x = static_cast<int>(ceil(double(Acols) / double(threadsPerBlock.x)));
        blocksPerGrid.y = static_cast<int>(ceil(double(Acols) / double(threadsPerBlock.y)));
    }

    matrixMultiplicationKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, Acols, Bcols);
    hipDeviceSynchronize();
}

void internalMatrixDoubleMultiply(double* A, double* B, double* C, int Acols, int Arows) {

    // declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block
    dim3 threadsPerBlock(Acols, Arows);
    dim3 blocksPerGrid(1, 1);
    if (Acols * Arows > 512) {
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
        blocksPerGrid.x = static_cast<int>(ceil(double(Acols) / double(threadsPerBlock.x)));
        blocksPerGrid.y = static_cast<int>(ceil(double(Arows) / double(threadsPerBlock.y)));
    }

    matrixMultiplicationKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, Acols, Arows);
    hipDeviceSynchronize();
}

void cuda_matrix_multiply(std::vector<double> A, std::vector<double> B, std::vector<double> C, int Acols, int Bcols) {

    cuda_array<double> d_A(A.size());
    cuda_array<double> d_B(B.size());
    cuda_array<double> d_C(C.size());

    d_A.set(A);
    d_B.set(B);

    internalMatrixMultiply(d_A.getData(), d_B.getData(), d_C.getData(), Acols, Bcols);
    d_C.get(C.data(), C.size());
}

void multiply_matrix_and_double(std::vector<double> A, double B, std::vector<double> C, int Acols, int Arows) {
    cuda_array<double> d_A(A.size());
    cuda_array<double> d_B(1);
    cuda_array<double> d_C(C.size());

    d_A.set(A);

    std::vector<double> bVector{ B };
    d_B.set(bVector);

    internalMatrixMultiply(d_A.getData(), d_B.getData(), d_C.getData(), Acols, Arows);
    d_C.get(C.data(), C.size());
}
