#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <vector>
#include <stdlib.h>
#include <stddef.h>
#include "CudaArray.h"
#include "Matrix_Operations.h"

using namespace std;

__global__ void matrixMaxKernel(double* A, double* B, double* C, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows && j < cols) {
        C[i * cols + j] = max(B[0], A[i * cols + j]);
    }
}

void internalMatrixMax(double* A, double* B, double* C, int rows, int cols) {

    // declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block
    dim3 threadsPerBlock(rows, cols);
    dim3 blocksPerGrid(1, 1);
    if (rows > 32) {
        threadsPerBlock.x = 32;
        blocksPerGrid.x = static_cast<int>(ceil(double(rows) / double(threadsPerBlock.x)));
    }

    if (cols > 32) {
        threadsPerBlock.y = 32;
        blocksPerGrid.y = static_cast<int>(ceil(double(cols) / double(threadsPerBlock.y)));
    }

    matrixMaxKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, rows, cols);
    hipDeviceSynchronize();
}

void matrix_max(const vector<double>& A, double input, vector<double>& C, int Arows, int Acols) {

    CudaArray<double> d_A(A.size());
    CudaArray<double> d_B(sizeof(double));
    CudaArray<double> d_C(C.size());

    d_A.set(A);
    d_B.set(vector<double>(input));

    internalMatrixMax(d_A.getData(), d_B.getData(), d_C.getData(), Arows, Acols);
    d_C.get(C.data(), C.size());
}
