#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cmath>
#include <vector>
#include "CudaArray.h"
#include "Matrix_Operations.h"

using namespace std;

__global__ static void matrixDoubleMultiplicationKernel(double* A, double* B, double* C, int Arows, int Acols) {
    int ROW = blockIdx.x * blockDim.x + threadIdx.x;
    int COL = blockIdx.y * blockDim.y + threadIdx.y;

    if (ROW < Arows && COL < Acols) {
        C[ROW * Acols + COL] = A[ROW * Acols + COL] * *B;
    }
}

static void internalMatrixDoubleMultiply(double* A, double* B, double* C, int Arows, int Acols) {

    // declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block
    dim3 threadsPerBlock(Arows, Acols);
    dim3 blocksPerGrid(1, 1);

    if (Arows > 32) {
        threadsPerBlock.x = 32;
        blocksPerGrid.x = static_cast<int>(ceil(double(Arows) / double(threadsPerBlock.x)));
    }

    if (Acols > 32) {
        threadsPerBlock.y = 32;
        blocksPerGrid.y = static_cast<int>(ceil(double(Acols) / double(threadsPerBlock.y)));
    }

    matrixDoubleMultiplicationKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, Arows, Acols);
    hipDeviceSynchronize();
}

void multiply_matrix_and_double(const vector<double>& A, const double B, vector<double>& C, int Arows, int Acols) {

    const std::vector<double> bVector{ B };

    CudaArray<double> d_A(A.size());
    CudaArray<double> d_B(bVector.size());
    CudaArray<double> d_C(C.size());

    d_A.set(A);
    d_B.set(bVector);

    internalMatrixDoubleMultiply(d_A.getData(), d_B.getData(), d_C.getData(), Arows, Acols);
    d_C.get(C.data(), C.size());
}