#include "hip/hip_runtime.h"
﻿#include "CudaAdd.h"

#include <stdio.h>


__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

CudaAdder::CudaAdder() {
    dev_a = 0;
    dev_b = 0; 
    dev_c = 0;
}

hipError_t CudaAdder::handleError() {
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    return cudaStatus;
}

hipError_t CudaAdder::allocateGPUMemory(int** dev, int size) {
    cudaStatus = hipMalloc((void**) dev, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return handleError();
    }
}

hipError_t CudaAdder::copyVectorToGPU(int* dev, const int* input, int size) {
    cudaStatus = hipMemcpy(dev, input, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return handleError();
    }
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t CudaAdder::addWithCuda(int* output, const int* input1, const int* input2, unsigned int size)
{
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return handleError();
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return handleError();
    }

    allocateGPUMemory(&dev_c, size);

    if (cudaStatus != hipSuccess) {
        return cudaStatus;
    }

    allocateGPUMemory(&dev_a, size);

    if (cudaStatus != hipSuccess) {
        return cudaStatus;
    }

    allocateGPUMemory(&dev_b, size);

    if (cudaStatus != hipSuccess) {
        return cudaStatus;
    }

    // Copy input vectors from host memory to GPU buffers.
    copyVectorToGPU(dev_a, input1, size);

    if (cudaStatus != hipSuccess) {
        return cudaStatus;
    }

    copyVectorToGPU(dev_b, input2, size);

    if (cudaStatus != hipSuccess) {
        return cudaStatus;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>> (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return handleError();
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        return handleError();
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(output, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return handleError();
    }

    return cudaStatus;
}